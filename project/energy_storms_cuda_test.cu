#include "hip/hip_runtime.h"
#include<sys/time.h>

/* Headers for the CUDA assignment versions */
#include<hip/hip_runtime.h>

#define TPB 256   // threads per block in x direction
#define RAD 1    // radius of stensil for hallo cells
/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */
#define CHECK_CUDA_CALL( a )	{ \
	hipError_t ok = a; \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}
#define CHECK_CUDA_LAST()	{ \
	hipError_t ok = hipGetLastError(); \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}


/* Use fopen function in local tests. The Tablon online judge software 
   substitutes it by a different function to run in its sandbox */
#ifdef CP_TABLON
#include "cputilstablon.h"
#else
#define    cp_open_file(name) fopen(name,"r")
#endif

/* Function to get wall time */
double cp_Wtime(){
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + 1.0e-6 * tv.tv_usec;
}


#define THRESHOLD    0.001f

/* Structure used to store data for one storm of particles */
typedef struct {
    int size;    // Number of particles
    int *posval; // Positions and values
} Storm;

/* THIS FUNCTION CAN BE MODIFIED */
/* Function to update a single position of the layer */
__device__ float update(int layer_size, int cp, int imp_pos, float energy ) {
    /* 1. Compute the absolute value of the distance between the
        impact position and the k-th position of the layer */
    int distance = imp_pos - cp;
    if ( distance < 0 ) distance = - distance;

    /* 2. Impact cell has a distance value of 1 */
    distance = distance + 1;

    /* 3. Square root of the distance */
    /* NOTE: Real world atenuation typically depends on the square of the distance.
       We use here a tailored equation that affects a much wider range of cells */
    float atenuacion = sqrtf( (float)distance );

    /* 4. Compute attenuated energy */
    float energy_k = energy / layer_size / atenuacion;

    /* 5. Do not add if its absolute value is lower than the threshold */
    if ( energy_k >= THRESHOLD / layer_size || energy_k <= -THRESHOLD / layer_size )
        return energy_k;
    }  else {
	    return 0.0f;
    }
}

/*Parallelezing the bombardment for loop */
__global__void bombardment(int storm_size, int layer_size, float *layer_d, int *posval_d){
  int cp = blockIdx.x * blockDim.x + threadIdx.x;

if ( cp < layer_size ) {

    float energy;
    int imp_pos;

    for (int j=  ; j < storm_size; j++ ) {
      energy = ((float)posval_d[ *j + ]) * 1000.0f;
      imp_pos = posval_d[ *j];
      layer_d[cp] += update(cp, layer_size, imp_pos, energy);

    }
  }
}
  

/*Parallelizing relaxation loop using shared memory*/
__global__ void relaxation(int layer_size, float *layer_d) {
  
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ float smem[TC + *RAD];

  if (i < layer_size) {

   int s_idx = threadIdx.x + RAD; 

   smem[s_idx] = layer_d[i];

   if (threadIdx.x < RAD){
     smem[s_idx - RAD] = layer_d[i- RAD];
     smem[s_idx + blockDim.x] = layer_d[i + blockDim.x];
   }

   __syncthreads();

   if (i != 0 && != layer_size -1)
     layer_d[i] = (smem[s_idx-1] + smem[s_idx] + smem[s_idx+1])/3;
  }
}




/* ANCILLARY FUNCTIONS: These are not called from the code section which is measured, leave untouched */
/* DEBUG function: Prints the layer status */
void debug_print(int layer_size, float *layer, int *positions, float *maximum, int num_storms ) {
    int i,k;
    /* Only print for array size up to 35 (change it for bigger sizes if needed) */
    if ( layer_size <= 35 ) {
        /* Traverse layer */
        for( k=0; k<layer_size; k++ ) {
            /* Print the energy value of the current cell */
            printf("%10.4f |", layer[k] );

            /* Compute the number of characters. 
               This number is normalized, the maximum level is depicted with 60 characters */
            int ticks = (int)( 60 * layer[k] / maximum[num_storms-1] );

            /* Print all characters except the last one */
            for (i=0; i<ticks-1; i++ ) printf("o");

            /* If the cell is a local maximum print a special trailing character */
            if ( k>0 && k<layer_size-1 && layer[k] > layer[k-1] && layer[k] > layer[k+1] )
                printf("x");
            else
                printf("o");

            /* If the cell is the maximum of any storm, print the storm mark */
            for (i=0; i<num_storms; i++) 
                if ( positions[i] == k ) printf(" M%d", i );

            /* Line feed */
            printf("\n");
        }
    }
}

/*
 * Function: Read data of particle storms from a file
 */
Storm read_storm_file( char *fname ) {
    FILE *fstorm = cp_open_file( fname );
    if ( fstorm == NULL ) {
        fprintf(stderr,"Error: Opening storm file %s\n", fname );
        exit( EXIT_FAILURE );
    }

    Storm storm;    
    int ok = fscanf(fstorm, "%d", &(storm.size) );
    if ( ok != 1 ) {
        fprintf(stderr,"Error: Reading size of storm file %s\n", fname );
        exit( EXIT_FAILURE );
    }

    storm.posval = (int *)malloc( sizeof(int) * storm.size * 2 );
    if ( storm.posval == NULL ) {
        fprintf(stderr,"Error: Allocating memory for storm file %s, with size %d\n", fname, storm.size );
        exit( EXIT_FAILURE );
    }
    
    int elem;
    for ( elem=0; elem<storm.size; elem++ ) {
        ok = fscanf(fstorm, "%d %d\n", 
                    &(storm.posval[elem*2]),
                    &(storm.posval[elem*2+1]) );
        if ( ok != 2 ) {
            fprintf(stderr,"Error: Reading element %d in storm file %s\n", elem, fname );
            exit( EXIT_FAILURE );
        }
    }
    fclose( fstorm );

    return storm;
}

/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
    int i,k;

    /* 1.1. Read arguments */
    if (argc<3) {
        fprintf(stderr,"Usage: %s <size> <storm_1_file> [ <storm_i_file> ] ... \n", argv[0] );
        exit( EXIT_FAILURE );
    }

    int layer_size = atoi( argv[1] );
    int num_storms = argc-2;
    Storm storms[ num_storms ];

    /* 1.2. Read storms information */
    for( i=2; i<argc; i++ ) 
        storms[i-2] = read_storm_file( argv[i] );

    /* 1.3. Intialize maximum levels to zero */
    float maximum[ num_storms ];
    int positions[ num_storms ];
    for (i=0; i<num_storms; i++) {
        maximum[i] = 0.0f;
        positions[i] = 0;
    }

    /* 2. Begin time measurement */
	CHECK_CUDA_CALL( hipSetDevice(0) );
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
    double ttotal = cp_Wtime();

    /* START: Do NOT optimize/parallelize the code of the main program above this point */

    /* 3. Allocate memory for the layer and initialize to zero */
    float *layer = (float *)malloc( sizeof(float) * layer_size );
    if ( layer == NULL ) {
        fprintf(stderr,"Error: Allocating the layer memory\n");
        exit( EXIT_FAILURE );
    }

    for( k=0; k<layer_size; k++ ) layer[k] = 0.0f;

/* Preliminary definitions for grid/block dimensions */
    dim3 blockDim(TC); 
    dim3 gridDim(ceil(((float)layer_size) / ((float)blockDim.x)));

    float *layer_d;
    int *posval_d;

    //Copy the data to GPU global memeory

    hipMalloc((void **)&smem, layer_size*sizeof(float));
    hipMemcpy(layer_d, layer, layer_size*sizeof(float), hipMemcpyHostToDevice);

-------------------------------------------------------------------------------------------------------|      hipMemcpy(layer_d, layer, layer_size*sizeof(float), hipMemcpyHostToDevice);

    /* 4. Storms simulation */
    for(int i=0; i<num_storms; i++) {

	// Allocate and copy the posval array onto the device 
      hipMalloc((void **)&posval_d        , 2 * storms[i].size * sizeof(int));	
      hipMemcpy(posval_d, storms[i].posval, 2 * storms[i].size * sizeof(int), hipMemcpyHostToDevice);
      
      /* Bombardment loop */
      bombardment<<<gridDim, blockDim>>>(storms[i].size, layer_size, layer_d, posval_d);      
      
      /* Relaxation loop*/
      relaxation<<<gridDim, blockDim>>>(layer_size, layer_d); 

      // Bring the layer array back to the host
      hipMemcpy(layer, layer_d, layer_size * sizeof(float), hipMemcpyDeviceToHost);

        /* 4.3. Locate the maximum value in the layer, and its position */
        for( k=1; k<layer_size-1; k++ ) {
            /* Check it only if it is a local maximum */
            if ( layer[k] > layer[k-1] && layer[k] > layer[k+1] ) {
                if ( layer[k] > maximum[i] ) {
                    maximum[i] = layer[k];
                    positions[i] = k;
                }
            }
        }
	hipFree(posval_d);
    }
     hipFree(layer_d);   
    /* END: Do NOT optimize/parallelize the code below this point */

    /* 5. End time measurement */
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
    ttotal = cp_Wtime() - ttotal;

    /* 6. DEBUG: Plot the result (only for layers up to 35 points) */
    #ifdef DEBUG
    debug_print( layer_size, layer, positions, maximum, num_storms );
    #endif

    /* 7. Results output, used by the Tablon online judge software */
    printf("\n");
    /* 7.1. Total computation time */
    printf("Time: %lf\n", ttotal );
    /* 7.2. Print the maximum levels */
    printf("Result:");
    for (i=0; i<num_storms; i++)
        printf(" %d %f", positions[i], maximum[i] );
    printf("\n");

    /* 8. Free resources */    
    for( i=0; i<argc-2; i++ )
        free( storms[i].posval );

    /* 9. Program ended successfully */
    return 0;
}

